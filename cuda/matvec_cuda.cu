#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gl_variables.h"
#include "gl_constants.h"

extern "C" int *matvec(double *alpha, double *x, double *beta, double *y);
extern "C" void comp_soleng_wrapper(double soleng);
extern "C" void comp_source_wrapper();
extern "C" void initGPU();
extern "C" void freeGPU();
__global__ void comp_pot(const double* xvct, double *atmchr, double *chrpos,
double *ptl, double *tr_xyz,double *tr_q, double *tr_area, int nface, int nchr);
__global__ void comp_source( double* bvct, double *atmchr, double *chrpos,
double *tr_xyz, double *tr_q, int nface, int nchr);

#define checkcudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}
// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

// get the ID from block
__global__ void initY( double* y, int nface) {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if(i<nface)
         y[i]=0.0;
}


__global__ void matvecmul(const double *x, double *y, double *q, int nface
        ,double *tr_xyz,double *tr_q, double *tr_area,double alpha, double beta){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j;
	double pre1,pre2;
	double area, rs, irs,sumrs;
	double G0, kappa_rs, exp_kappa_rs, Gk;
	double cos_theta, cos_theta0, tp1, tp2, dot_tqsq;
	double G10, G20, G1, G2, G3, G4;
	double L1, L2, L3, L4;

    pre1=0.50*(1.0+eps); /* eps=80.0 a constant */
    pre2=0.50*(1.0+fdivide(1.0,eps)); //!!! fdivide?
    if (i < nface) {

		double3 tp={tr_xyz[3*i],tr_xyz[3*i+1],tr_xyz[3*i+2]};
		double3 tq={tr_q[3*i],tr_q[3*i+1],tr_q[3*i+2]};

		double2 peng={0.0,0.0};
		for (j=0;j<nface;j++){
          if (j != i){
			double3 sp={tr_xyz[3*j],tr_xyz[3*j+1],tr_xyz[3*j+2]};
			double3 sq={tr_q[3*j],tr_q[3*j+1],tr_q[3*j+2]};
			double3 r_s={sp.x-tp.x,sp.y-tp.y,sp.z-tp.z};
			sumrs= r_s.x*r_s.x + r_s.y*r_s.y+r_s.z*r_s.z;
			rs=sqrt(sumrs);
			irs=rsqrt(sumrs);
			G0=one_over_4pi;
			G0=G0*irs;
			kappa_rs=kappa*rs;
			exp_kappa_rs=exp(-kappa_rs);
			Gk=exp_kappa_rs*G0;

			cos_theta	=(sq.x*r_s.x+sq.y*r_s.y+sq.z*r_s.z)*irs;
			cos_theta0	=(tq.x*r_s.x+tq.y*r_s.y+tq.z*r_s.z)*irs;

			tp1=G0*irs;
			tp2=(1.0+kappa_rs)*exp_kappa_rs;

			G10=cos_theta0*tp1;
			G20=tp2*G10;

			G1=cos_theta*tp1;
			G2=tp2*G1;

			dot_tqsq=sq.x*tq.x+sq.y*tq.y+sq.z*tq.z;
			G3=(dot_tqsq-3.0*cos_theta0*cos_theta)*irs*tp1;
			G4=tp2*G3-kappa2*cos_theta0*cos_theta*Gk;
			L1=G1-eps*G2;
			L2=G0-Gk;
			L3=G4-G3;
			L4=G10-fdivide(G20,eps);

			double2 peng_old={x[j], x[j+nface]};
			area=tr_area[j];
			peng.x=peng.x+(L1*peng_old.x+L2*peng_old.y)*area;
			peng.y=peng.y+(L3*peng_old.x+L4*peng_old.y)*area;
          }
		}

		y[i]=y[i] *beta+(pre1*x[i]-peng.x) * alpha;
		y[nface+i]=y[nface+i] * beta+(pre2*x[nface+i]-peng.y) * alpha;
	}
}

double *d_X, *d_Y,*d_tr_xyz,*d_tr_q,*d_tr_area,*d_atmchr,*d_chrpos,*d_chrptl,*d_xvct;
int threadsPerBlock = 256;

void initGPU() {
	checkcudaErrors(hipMalloc((void**)&d_X,2*nface*sizeof(double))) ;
    checkcudaErrors(hipMalloc((void**)&d_Y,2*nface*sizeof(double))) ;
    checkcudaErrors(hipMalloc((void**)&d_tr_q,3*nface*sizeof(double))) ;
    checkcudaErrors(hipMalloc((void**)&d_tr_xyz,3*nface*sizeof(double))) ;
    checkcudaErrors(hipMalloc((void**)&d_tr_area,nface*sizeof(double))) ;
    checkcudaErrors(hipMalloc((void**)&d_atmchr,nchr*sizeof(double))) ;
    checkcudaErrors(hipMalloc((void**)&d_chrpos,3*nchr*sizeof(double))) ;

    checkcudaErrors(hipMemcpy(d_tr_area,tr_area,nface*sizeof(double),hipMemcpyHostToDevice));
    checkcudaErrors(hipMemcpy(d_tr_xyz,tr_xyz,3*nface*sizeof(double),hipMemcpyHostToDevice));
    checkcudaErrors(hipMemcpy(d_tr_q,tr_q,3*nface*sizeof(double),hipMemcpyHostToDevice));
    checkcudaErrors(hipMemcpy(d_atmchr,atmchr,nchr*sizeof(double),hipMemcpyHostToDevice));
    checkcudaErrors(hipMemcpy(d_chrpos,chrpos,3*nchr*sizeof(double),hipMemcpyHostToDevice));
}


void freeGPU() {
    checkcudaErrors( hipFree(d_tr_area));
    checkcudaErrors( hipFree(d_tr_xyz));
    checkcudaErrors( hipFree(d_tr_q));
    checkcudaErrors( hipFree(d_X));
    checkcudaErrors( hipFree(d_Y));
    checkcudaErrors( hipFree(d_xvct));
    checkcudaErrors( hipFree(d_atmchr));
    checkcudaErrors( hipFree(d_chrpos));
    checkcudaErrors( hipFree(d_chrptl));
}

/* This subroutine wraps the matrix-vector multiplication */
int *matvec(double *alpha, double *x, double *beta, double *y) {
	int blocksPerGrid = (nface + threadsPerBlock - 1) / threadsPerBlock;
    checkcudaErrors(hipMemcpy(d_X, x,2*nface*sizeof(double), hipMemcpyHostToDevice));
    checkcudaErrors(hipMemcpy(d_Y, y,2*nface*sizeof(double), hipMemcpyHostToDevice));
    matvecmul<<<blocksPerGrid, threadsPerBlock>>>(d_X, d_Y,d_tr_q,nface
            ,d_tr_xyz,d_tr_q,d_tr_area, *alpha, *beta);
    getLastCudaError("kernel launch failure");
    checkcudaErrors(hipMemcpy(y, d_Y, 2*nface*sizeof(double), hipMemcpyDeviceToHost));
    return NULL;
}

/* This subroutine wraps the solvation energy computation */
/* Called before freeGPU() */
void comp_soleng_wrapper(double soleng) {
    int i;
	double *chrptl;
	int blocksPerGrid = (nface + threadsPerBlock - 1) / threadsPerBlock;
	double units_para=2.0;
    units_para=units_para*units_coef;
    units_para=units_para*pi;

	if ((chrptl=(double *) malloc(nface*sizeof(double)))==NULL){
		printf("error in allcating chrptl");
	}
	checkcudaErrors(hipMalloc((void**)&d_chrptl,nface*sizeof(double))) ;
	checkcudaErrors(hipMalloc((void**)&d_xvct,2*nface*sizeof(double))) ;
    checkcudaErrors(hipMemcpy(d_xvct,xvct,2*nface*sizeof(double),hipMemcpyHostToDevice));

	comp_pot<<<blocksPerGrid, threadsPerBlock>>>(d_xvct, d_atmchr, d_chrpos,
    d_chrptl, d_tr_xyz,d_tr_q, d_tr_area, nface, nchr);
    checkcudaErrors(hipMemcpy(chrptl,d_chrptl,nface*sizeof(double),hipMemcpyDeviceToHost));

	soleng=0.0;
	for (i=0;i<nface;i++) soleng=soleng+chrptl[i];
	soleng=soleng*units_para;
	printf("solvation energy on GPU = %f kcal/mol\n",soleng);
}



/* This subroutine calculates the element-wise potential on GPU */
__global__ void comp_pot(const double* xvct, double *atmchr, double *chrpos,
double *ptl, double *tr_xyz,double *tr_q, double *tr_area, int nface, int nchr){

	int j = blockDim.x * blockIdx.x + threadIdx.x;
    double sumrs,irs,rs,G0,Gk,kappa_rs,exp_kappa_rs;
    double cos_theta,G1,G2,L1,L2,tp1,tp2;
    int i;
	if (j<nface){
    	ptl[j]=0.0;
		double3 r={tr_xyz[3*j],tr_xyz[3*j+1],tr_xyz[3*j+2]};
		double3 v={tr_q[3*j],tr_q[3*j+1],tr_q[3*j+2]};
    	for (i=0; i<nchr; i++) {
        	double3 s={chrpos[3*i],chrpos[3*i+1],chrpos[3*i+2]};
			double3 r_s={r.x-s.x,r.y-s.y,r.z-s.z};
			sumrs= r_s.x*r_s.x + r_s.y*r_s.y+r_s.z*r_s.z;
			rs=sqrt(sumrs);
			irs=rsqrt(sumrs);

        	G0=one_over_4pi;
        	G0=G0*irs;
        	kappa_rs=kappa*rs;
        	exp_kappa_rs=exp(-kappa_rs);
        	Gk=exp_kappa_rs*G0;

        	cos_theta=(v.x*r_s.x+v.y*r_s.y+v.z*r_s.z)*irs;

        	tp1=G0*irs;
        	tp2=(1.0+kappa_rs)*exp_kappa_rs;

        	G1=cos_theta*tp1;
        	G2=tp2*G1;

        	L1=G1-eps*G2;
        	L2=G0-Gk;

      		ptl[j]=ptl[j]+atmchr[i]*(L1*xvct[j]+L2*xvct[nface+j])*tr_area[j];
		}
    }
}

/* This subroutine wraps the solvation energy computation */
/* In main_cuda.c after initGPU() */
void comp_source_wrapper() {
	double *d_bvct;
    int blocksPerGrid = (nface + threadsPerBlock - 1) / threadsPerBlock;

	checkcudaErrors(hipMalloc((void**)&d_bvct,2*nface*sizeof(double))) ;

	comp_source<<<blocksPerGrid, threadsPerBlock>>>(d_bvct, d_atmchr, d_chrpos,
    d_tr_xyz,d_tr_q, nface, nchr);
    checkcudaErrors(hipMemcpy(bvct,d_bvct,2*nface*sizeof(double),hipMemcpyDeviceToHost));
    checkcudaErrors(hipFree(d_bvct));
}


/* This subroutine calculates the source term of the integral equation on GPU */
/* atmchr=atom charge   chrpos=charge position */
/* bvct be located at readin.c */
__global__ void comp_source( double* bvct, double *atmchr, double *chrpos,
double *tr_xyz,double *tr_q, int nface, int nchr){

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j;
	double sumrs,cos_theta,irs,G0,G1,tp1;
	if (i<nface){
        bvct[i]=0.0;
        bvct[i+nface]=0.0;
        for (j=0; j<nchr; j++){
            double3 r_s={	chrpos[3*j]-tr_xyz[3*i],
							chrpos[3*j+1]-tr_xyz[3*i+1],
							chrpos[3*j+2]-tr_xyz[3*i+2]};
			sumrs= r_s.x*r_s.x + r_s.y*r_s.y+r_s.z*r_s.z; //c can't use that r_s.x
            cos_theta=tr_q[3*i]*r_s.x+tr_q[3*i+1]*r_s.y+tr_q[3*i+2]*r_s.z;
			irs=rsqrt(sumrs);//returns reciprocal square root of scalars and vectors.
            cos_theta=cos_theta*irs;
            G0=one_over_4pi;//constant
            G0=G0*irs;
            tp1=G0*irs;
            G1=cos_theta*tp1;
            bvct[i]=bvct[i]+atmchr[j]*G0;
            bvct[nface+i]=bvct[nface+i]+atmchr[j]*G1;
        }

    }
}
